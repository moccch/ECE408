// vector_add.cu


#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

const int N = 1 << 20; // 1 million elements

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    int size = N * sizeof(int);

    a = new int[N];
    b = new int[N];
    c = new int[N];

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<(N+255)/256, 256>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        assert(c[i] == a[i] + b[i]);
    }

    std::cout << "CUDA Vector Addition Successful!" << std::endl;

    delete[] a;
    delete[] b;
    delete[] c;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
